#include "hip/hip_runtime.h"
#include "Lambertian.h"
#include "../Utils/MathsCommons.h"



LambertianBSDF::LambertianBSDF(const Color& baseColor_):baseColor(baseColor_){

}

Color LambertianBSDF::eval(float3 incident, float3 exitant){
    return baseColor / M_PI;
}
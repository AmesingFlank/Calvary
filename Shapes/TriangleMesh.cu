#include "hip/hip_runtime.h"
#include "TriangleMesh.h"
#include <iostream>
#include <happly.h>

__host__ 
TriangleMesh::TriangleMesh():
trianglesCount(0),
verticesCount(0),
hasVertexNormals(0),
positions(0,true),
normals(0,true),
texCoords(0,true),
indices(0,true)
{
    std::cout << "default constructing trianglemesh" << std::endl;
}

__host__ 
TriangleMesh::TriangleMesh(int trianglesCount_, int verticesCount_,bool hasVertexNormals_, bool isCopyForKernel_):
trianglesCount(trianglesCount_),
verticesCount(verticesCount_),
hasVertexNormals(hasVertexNormals_),
positions(verticesCount_,isCopyForKernel_),
normals(verticesCount_,isCopyForKernel_),
texCoords(verticesCount,isCopyForKernel_),
indices(trianglesCount,isCopyForKernel_)
{

}


TriangleMesh TriangleMesh::getCopyForKernel(){
    TriangleMesh copy(trianglesCount,verticesCount,hasVertexNormals,true);
    copy.positions.gpu.data = positions.gpu.data;
    copy.normals.gpu.data = normals.gpu.data;
    copy.texCoords.gpu.data = texCoords.gpu.data;
    copy.indices.gpu.data = indices.gpu.data;
    return copy;
}

void TriangleMesh::copyToDevice() {
    positions.copyToDevice();
    normals.copyToDevice();
    texCoords.copyToDevice();
    indices.copyToDevice();
}


TriangleMesh TriangleMesh::createFromPLY(const std::string& filename,const glm::mat4& transform){
    happly::PLYData plyIn(filename);

    std::vector<float> positionsX = plyIn.getElement("vertex").getProperty<float>("x");
    std::vector<float> positionsY= plyIn.getElement("vertex").getProperty<float>("y");
    std::vector<float> positionsZ= plyIn.getElement("vertex").getProperty<float>("z");

    std::vector<std::vector<int>> indices = 
        plyIn.getElement("face").getListProperty<int>("vertex_indices");

    int trianglesCount = indices.size();
    int verticesCount = positionsX.size();
    TriangleMesh mesh(trianglesCount,verticesCount,false,false);

    for(int i = 0;i<verticesCount;++i){
        float3 pos = make_float3(positionsX[i],positionsY[i],positionsZ[i]);
        pos = to_float3(transform * glm::vec4(to_vec3(pos), 1.f));
        mesh.positions.cpu.data[i] = pos;
    }

    for(int i = 0;i<trianglesCount;++i){
        int3 thisIndices = make_int3(indices[i][0],indices[i][1],indices[i][2]);
        mesh.indices.cpu.data[i] = thisIndices;
    }

    mesh.copyToDevice();
    return mesh;

}


TriangleMesh TriangleMesh::createFromParams(const Parameters& params,const glm::mat4& transform,const std::filesystem::path& basePath){
    
    std::string plyPathString = params.getString("filename");
    std::filesystem::path plyRelativePath(plyPathString);
    std::string filename = (basePath/plyRelativePath).generic_string();
    
    return TriangleMesh::createFromPLY(filename,transform);

}
#include "hip/hip_runtime.h"
#include "Parsing.h"
#include "../Utils/MathsCommons.h"
#include "../Core/Material.h"
#include "../Core/Primitive.h"
#include "../Samplers/SimpleSampler.h"
#include "../Integrators/DirectLightingIntegrator.h"
#include "../Integrators/PathTracingIntegrator.h"
#include "../Integrators/CreateIntegrator.h"
#include "../Utils/Utils.h"
#include <unordered_map>
#include <string>
#include "Lexing.h"
#include "../Core/Texture.h"
#include "../Core/Impl.h"

#define SIGNAL_PARSING_ERROR(err,pos,tokenString) SIGNAL_ERROR((std::string("Parsing Error: ")+err+std::string("\n at token ")+std::to_string(pos)+": "+tokenString).c_str())

template<typename T>
struct NamedStorage {
	std::unordered_map<std::string, T> items;
	bool has(const std::string& name) {
		return items.find(name) != items.end();
	}
	void add(const std::string& name, const T& item) {
		items.insert({ name, item });
	}
	T get(const std::string& name) {
		if (!has(name)) {
			SIGNAL_ERROR((std::string("Named T not found :")+name).c_str());
		}
		return items.at(name);
	}
};
using MaterialStorage = NamedStorage<MaterialObject>;
using TextureStorage = NamedStorage<Texture2D>;


std::vector<float> readNumList(TokenBuf& buf){
	std::vector<float> result;
	buf.checkAndPop<LeftSquareBracketToken>();
	while(buf.peek()->type == TokenType::Num){
		std::shared_ptr<NumToken> num = buf.checkAndPop<NumToken>();
		result.push_back(num->value);
		
	}
	buf.checkAndPop<RightSquareBracketToken>();
	return result;
}

std::vector<std::string> readStringList(TokenBuf& buf) {
	std::vector<std::string> result;
	buf.checkAndPop<LeftSquareBracketToken>();
	while (buf.peek()->type == TokenType::String) {
		std::shared_ptr<StringToken> s = buf.checkAndPop<StringToken>();
		result.push_back(s->all);

	}
	buf.checkAndPop<RightSquareBracketToken>();
	return result;
}

void readUntilNextKeyWorkd(TokenBuf& buf) {
	buf.checkAndPop<KeyWordToken>();
	while (buf.peek()->type != TokenType::KeyWord) {
		buf.moveForward();
	}
}

ObjectDefinition readObjectDefinition(TokenBuf& buf){
	ObjectDefinition def;

	std::shared_ptr<KeyWordToken> keyWord = buf.checkAndPop<KeyWordToken>();
	def.keyWord = keyWord -> word;

	std::shared_ptr<StringToken> name = buf.checkAndPop<StringToken>();
	def.objectName = name->words[0];

	while(buf.peek()->type == TokenType::String){
		std::shared_ptr<StringToken> key = buf.checkAndPop<StringToken>();
		std::string fieldName = key->words[key->words.size()-1];

		auto nextToken = buf.peek();
		switch(nextToken->type){
			case TokenType::String:
				def.params.strings[fieldName] = buf.checkAndPop<StringToken>()->all;
				break;
			case TokenType::Num:
				def.params.nums[fieldName] = buf.checkAndPop<NumToken>()->value;
				break;
			case TokenType::LeftSquareBracket:
				if (buf.peek(1)->type == TokenType::Num) {
					def.params.numLists[fieldName] = readNumList(buf);
				}
				else if (buf.peek(1)->type == TokenType::String) {
					def.params.stringLists[fieldName] = readStringList(buf);
				}
				else {
					SIGNAL_ERROR((std::string("Unaccepted List Element. Token index: ") + std::to_string(buf.currentIndex+1) + "." + buf.peek(1)->print()).c_str());
				}
				break;
			default:
				SIGNAL_ERROR((std::string("Read Object failed. Token index: ")+std::to_string(buf.currentIndex)+ "." + nextToken->print()).c_str());
				break;
		}
	}
	def.isDefined = true;
	return def;
}



void readLookAt(TokenBuf& buf, float3& eye, float3& center, float3& up){
	auto lookAt = buf.checkAndPop<KeyWordToken>();
	if(lookAt->word != "LookAt"){
		SIGNAL_ERROR("LookAt not found when calling readLookAt.");
	}
	eye.x = buf.checkAndPop<NumToken>()->value;
	eye.y = buf.checkAndPop<NumToken>()->value;
	eye.z = buf.checkAndPop<NumToken>()->value;

	center.x = buf.checkAndPop<NumToken>()->value;
	center.y = buf.checkAndPop<NumToken>()->value;
	center.z = buf.checkAndPop<NumToken>()->value;

	up.x = buf.checkAndPop<NumToken>()->value;
	up.y = buf.checkAndPop<NumToken>()->value;
	up.z = buf.checkAndPop<NumToken>()->value;
}


bool readTransform(TokenBuf& buf, glm::mat4& transform){
	auto nextToken = buf.peek();
	auto keyWord = std::dynamic_pointer_cast<KeyWordToken>(nextToken);
	if(keyWord){
		std::string word = keyWord->word;
		if(word=="Translate"){
			buf.moveForward();
			float x = buf.checkAndPop<NumToken>()->value;
			float y = buf.checkAndPop<NumToken>()->value;
			float z = buf.checkAndPop<NumToken>()->value;
			transform = glm::translate(transform,glm::vec3(x,y,z));
			return true;
		}
		if(word=="Rotate"){
			buf.moveForward();
			float angle = buf.checkAndPop<NumToken>()->value;
			float x = buf.checkAndPop<NumToken>()->value;
			float y = buf.checkAndPop<NumToken>()->value;
			float z = buf.checkAndPop<NumToken>()->value;
			transform = glm::rotate(transform,glm::radians(angle),glm::vec3(x,y,z));
			return true;
		}
		if(word=="Scale"){
			buf.moveForward();
			float x = buf.checkAndPop<NumToken>()->value;
			float y = buf.checkAndPop<NumToken>()->value;
			float z = buf.checkAndPop<NumToken>()->value;
			transform = glm::scale(transform,glm::vec3(x,y,z));
			return true;
		}
		if (word == "Transform") {
			buf.moveForward();
			std::vector<float> matData = readNumList(buf);
			glm::mat4 mat = to_mat4(matData);
			transform = mat * transform;
			return true;
		}

		else{
			return false;
		}
	}
	SIGNAL_PARSING_ERROR("Keyword expected.",buf.currentIndex,nextToken->print());
}


void parseSceneWideOptions(TokenBuf& buf,RenderSetup& result){

	glm::mat4 transform(1.0);

	ObjectDefinition cameraDef;
	ObjectDefinition filmDef;
	ObjectDefinition integratorDef;
	ObjectDefinition samplerDef;

	// parse scene-wide options
	while(true){
		auto nextToken = buf.peek();
		auto keyWord = std::dynamic_pointer_cast<KeyWordToken>(nextToken);
		if(keyWord){
			if(keyWord->word == "WorldBegin"){
				break;
			}
			else if(keyWord->word == "LookAt"){
				float3 eye,center,up;
				
				readLookAt(buf,eye,center,up);
				eye = apply(transform, eye);
				center = apply(transform, center);
				up = glm::mat3(transform) * up;
				transform = glm::lookAtLH(to_vec3(eye), to_vec3(center), to_vec3(up)) * transform;
			}
			else if(keyWord->word == "Camera"){
				cameraDef = readObjectDefinition(buf);
			}
			else if(keyWord->word == "Film"){
				filmDef = readObjectDefinition(buf);
			}
			else if(keyWord->word == "Sampler"){
				samplerDef = readObjectDefinition(buf);
			}
			else if (readTransform(buf, transform)) {

			}
			else if(keyWord->word == "Integrator"){
				integratorDef = readObjectDefinition(buf);
			}
			
			else{
				std::cout<<"reading unrecognized object from "<<buf.currentIndex;
				readUntilNextKeyWorkd(buf);
				std::cout<<"done"<<std::endl;
			}
		}
		else{
			SIGNAL_PARSING_ERROR("Keyword expected.",buf.currentIndex,nextToken->print());
		}
	}

	if(!(cameraDef.isDefined && filmDef.isDefined && integratorDef.isDefined && samplerDef.isDefined)){
		SIGNAL_ERROR("incomplete scene-wide options");
	}

	auto integrator = CreateIntegrator::createFromObjectDefinition(integratorDef);
	integrator->sampler = std::make_unique<SamplerObject>(SamplerObject::createFromObjectDefinition(samplerDef));

	result.renderer.integrator = std::move(integrator);
	result.renderer.film = std::make_unique<FilmObject>(FilmObject::createFromObjectDefinition(filmDef));
	int width = result.renderer.film->getWidth();
	int height = result.renderer.film->getHeight();
	result.renderer.camera = std::make_unique<CameraObject>(CameraObject::createFromObjectDefinition(cameraDef,glm::inverse(transform),width,height));

	if (filmDef.params.hasString("filename")) {
		result.outputFileName = filmDef.params.getString("filename");
	}
	else {
		result.outputFileName = "output.png";
	}

}



void parseSubsection(TokenBuf& buf, RenderSetup& result, glm::mat4 transform,const std::filesystem::path& basePath, MaterialStorage& materialsStore,TextureStorage& textureStore, bool& nextShapeHasAreaLight,std::shared_ptr<MaterialObject> currentMaterial) {
	auto begin = buf.checkAndPop<KeyWordToken>();
	if ( !endsWith(begin->word,"Begin") ) {
		SIGNAL_PARSING_ERROR("XXXBegin expected.", buf.currentIndex, begin->print());
	}

	std::string subsectionName = begin->word.substr(0, begin->word.size() - std::string("Begin").size());

	
	bool shouldReverseOrientation = false;

	while (true) {
		auto nextToken = buf.peek();
		auto keyWord = std::dynamic_pointer_cast<KeyWordToken>(nextToken);
		if (keyWord) {
			if (endsWith(keyWord->word, "End")) {
				std::string endingSubsectionName = keyWord->word.substr(0, keyWord->word.size() - std::string("End").size());
				if (endingSubsectionName != subsectionName) {
					SIGNAL_PARSING_ERROR("Mismatching subsection names.", buf.currentIndex, nextToken->print());
				}
				break;
			}
			else if (endsWith(keyWord->word,"Begin")) {
				parseSubsection(buf, result, transform, basePath,materialsStore,textureStore,nextShapeHasAreaLight,currentMaterial);
			}
			else if (keyWord->word == "Shape") {
				auto shapeDef = readObjectDefinition(buf);
				TriangleMesh shape = TriangleMesh::createFromObjectDefinition(shapeDef, transform, basePath);
				if (shouldReverseOrientation) {
					shape.reverseOrientation = true;
				}
				
				Primitive prim;
				
				prim.shape = shape;

				if (currentMaterial.get() != nullptr) {
					prim.material = *currentMaterial;
				}
				else {
					MaterialObject  matteGray = MatteMaterial(make_float3(1, 1, 1));
					prim.material = matteGray;
				}

				if (nextShapeHasAreaLight) {
					prim.setAreaLightIndex(result.scene.lightsHost.size() - 1);
					nextShapeHasAreaLight = false;
				}

				result.scene.primitivesHost.push_back(prim);
			}
			else if (keyWord->word == "AreaLightSource") {
				auto lightDef = readObjectDefinition(buf);
				LightObject light = LightObject::createFromObjectDefinition(lightDef, transform);
				if (light.is<DiffuseAreaLight>()) {
					DiffuseAreaLight* diffuseLight = light.get<DiffuseAreaLight>();
					// shapeIndex will be the index of the next shape to be added.
					diffuseLight->shapeIndex = result.scene.primitivesHost.size();
					std::cout << "added shape for diffuse area light "<< diffuseLight->shapeIndex << std::endl;
					nextShapeHasAreaLight = true;
				}

				result.scene.lightsHost.push_back(light);

			}
			else if (keyWord->word == "LightSource") {
				auto lightDef = readObjectDefinition(buf);
				LightObject light = LightObject::createFromObjectDefinition(lightDef, transform);
				result.scene.lightsHost.push_back(light);
				if (lightDef.objectName == "infinite") {
					result.scene.environmentMapIndex = result.scene.lightsHost.size() - 1;
				}
			}
			else if (readTransform(buf, transform)) {

			}
			else if (keyWord->word == "MakeNamedMaterial") {
				auto def = readObjectDefinition(buf);
				materialsStore.add(def.objectName,MaterialObject::createFromObjectDefinition(def,textureStore.items));
			}
			else if (keyWord->word == "Material") {
				auto def = readObjectDefinition(buf);
				def.params.strings["type"] = def.objectName;
				currentMaterial = std::make_shared<MaterialObject>(MaterialObject::createFromObjectDefinition(def, textureStore.items));
			}
			else if (keyWord->word == "Texture") {
				auto def = readObjectDefinition(buf);
				textureStore.add(def.objectName,Texture2D::createFromObjectDefinition(def,transform,basePath));
			}
			else if (keyWord->word == "NamedMaterial") {
				buf.moveForward();
				std::string name = buf.checkAndPop<StringToken>()->all;
				currentMaterial = std::make_shared<MaterialObject>(materialsStore.get(name));
			}
			else if (keyWord->word == "Include") {
				buf.moveForward();
				std::string includeFile = buf.checkAndPop<StringToken>()->all;
				std::filesystem::path includePath = basePath / std::filesystem::path(includeFile);
				TokenBuf includedTokens = runLexing(includePath);
				buf.insertHere(includedTokens);
			}
			else if (keyWord->word == "ReverseOrientation") {
				buf.moveForward();
				shouldReverseOrientation = true;
			}
			else {
				std::cout << "reading unrecognized object:" << keyWord->word<< " from " << buf.currentIndex ;
				readUntilNextKeyWorkd(buf);
				std::cout << "done" << std::endl;
			}
		}
		else {
			SIGNAL_PARSING_ERROR("Keyword expected.", buf.currentIndex, nextToken->print());
		}
	}

	buf.checkAndPop<KeyWordToken>();
}


RenderSetup runParsing(TokenBuf tokens, const std::filesystem::path& basePath) {

	RenderSetup result;
	MaterialStorage materials;
	TextureStorage textures;


	parseSceneWideOptions(tokens, result);

	bool nextShapeHasAreaLight = false;
	std::shared_ptr<MaterialObject> currentMaterial = nullptr;
	glm::mat4 currentTransform(1.0);

	parseSubsection(tokens,result,currentTransform,basePath,materials,textures,nextShapeHasAreaLight,currentMaterial);

	return result;
}
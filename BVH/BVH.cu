#include "hip/hip_runtime.h"
#include "../Utils/GpuCommons.h"
#include "../Utils/MathsCommons.h"


#include "BVH.h"
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

// do not change
#define MORTEN_BITS_PER_DIMENSION 10



BVH::BVH():primitivesCount(0),nodes(0,true){

}

BVH::BVH(int primitivesCount_,bool isCopyForKernel_):primitivesCount(primitivesCount_),nodes(primitivesCount_ * 2 - 1,isCopyForKernel_){

}

BVH BVH::getCopyForKernel(){
    BVH copy(primitivesCount,true);
    copy.nodes.gpu.data = nodes.gpu.data;
    return copy;
}

struct BVHLeafNode{
    AABB box;
    int primitiveIndex;
    int parent;
};

struct BVHInternalNode{
    AABB box;
    int leftChild;
    int rightChild;
    int parent;
    unsigned int visited;
    bool leftChildIsLeaf;
    bool rightChildIsLeaf;
};


__global__
void fillLeafBoundingBoxes(Triangle* primitivesDevice, int primitivesCount,BVHLeafNode* nodes ){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= primitivesCount) return;

    nodes[index].box = primitivesDevice[index].getBoundingBox();
    nodes[index].primitiveIndex = index;
}


__host__ __device__
unsigned int shiftMorton(unsigned int x){
    if (x >= (1 << MORTEN_BITS_PER_DIMENSION)) 
        x= 1 << MORTEN_BITS_PER_DIMENSION-1;
    x = (x | (x << 16)) & 0b00000011000000000000000011111111;
    x = (x | (x << 8)) & 0b00000011000000001111000000001111;
    x = (x | (x << 4)) & 0b00000011000011000011000011000011;
    x = (x | (x << 2)) & 0b00001001001001001001001001001001;
    return x;
};


__global__
void fillLeafMortonCodes(Triangle* primitivesDevice, int primitivesCount,BVHLeafNode* leaves,unsigned int* leafMortonCodes,AABB sceneBounds ){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= primitivesCount) return;

    float3 centroid = (leaves[index].box.centroid() - sceneBounds.centroid()) / sceneBounds.extent();
    int factor = 1<<MORTEN_BITS_PER_DIMENSION;
    unsigned int x = centroid.x * factor;
    unsigned int y = centroid.y * factor;
    unsigned int z = centroid.z * factor;

    unsigned int code = (shiftMorton(x) << 2) | (shiftMorton(y) << 1) | shiftMorton(z);
    leafMortonCodes[index] = code;
}



__host__ __device__
int sign(int val) {
    return (val >= 0)?1:-1;
}

__host__ __device__
int commonPrefixLength(unsigned int* mortonCodes, unsigned int i,unsigned int j){
    unsigned int mortonI = mortonCodes[i];
    unsigned int mortonJ = mortonCodes[j];
    if(mortonI != mortonJ){
        for(int bit = 3*MORTEN_BITS_PER_DIMENSION-1; bit>=0;--bit){
            if(((mortonI >> bit) & 1) != ((mortonJ >> bit) & 1)){
                return 3*MORTEN_BITS_PER_DIMENSION-1 - bit;
            }
        }
    }

    for(int bit = 31; bit>=0;--bit){
        if(((i >> bit) & 1) != ((j >> bit) & 1)){
            return (31 - bit) + 3*MORTEN_BITS_PER_DIMENSION;
        }
    }
    
    return 32+3*MORTEN_BITS_PER_DIMENSION; // shuoldn't happen
}

__global__ 
void buildRadixTree(int leavesCount,unsigned int* leafMortonCodes, BVHLeafNode* leaves, BVHInternalNode* internals){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= leavesCount - 1) return;

    int d = 1;
    if(i > 0){
        d = sign(commonPrefixLength(leafMortonCodes,i,i+1) - commonPrefixLength(leafMortonCodes,i,i-1));
    }

    int minCommonPrefixLength = 0;
    if(i > 0){
        minCommonPrefixLength = commonPrefixLength(leafMortonCodes,i,i-d);
    }

    unsigned int maxLength = 2;

    while( i+ maxLength*d >=0 && i+maxLength*d < leavesCount && commonPrefixLength(leafMortonCodes,i,i+maxLength*d) > minCommonPrefixLength){
        maxLength *= 2;
    }
    

    unsigned int length = 0;
    for(int power = 1;; ++power){
        unsigned int t = maxLength >> power;

        if(t==0){
            break;
        }
        int j = i+(length+t)*d;
        if( j >= 0 && j <leavesCount && commonPrefixLength(leafMortonCodes,i,j) > minCommonPrefixLength){
            length += t;
        }
    }

    int j = i + length *  d; // other end;
    int prefixLength = commonPrefixLength(leafMortonCodes,i,j);

    if (i == 0) {
        j = leavesCount - 1;
        prefixLength = 0;
    }


    unsigned int distanceToSplit = 0;
    for(int power = 1;; ++power){
        unsigned int t = ceil( (float)length / ((float) (1<<power)) );
        
        int j = i + (distanceToSplit + t)*d;
        if(j >= 0 && j <leavesCount && commonPrefixLength(leafMortonCodes,i,j) > prefixLength){
            distanceToSplit += t;
        }

        if (t <= 1) {
            break;
        }
    }

    

    int splitPos = i + distanceToSplit*d + min(0,d);

    int leftChild = splitPos;
    int rightChild = splitPos + 1;

    internals[i].leftChild = leftChild;
    internals[i].rightChild = rightChild;

    internals[i].leftChildIsLeaf = min(i,j)==leftChild;
    internals[i].rightChildIsLeaf = (max(i,j) == rightChild) || (rightChild == leavesCount - 1);

    if(internals[i].leftChildIsLeaf){
        leaves[leftChild].parent = i;
    }
    else{
        internals[leftChild].parent = i;
    }

    if(internals[i].rightChildIsLeaf){
        leaves[rightChild].parent = i;
    }
    else{
        internals[rightChild].parent = i;
    }
    /*
    if (i >= 0) {
        printf("i:%d,\t j:%d,\t d:%d,\t splitPos: %d,\t minDelta:%d,\t delta:%d,\t maxLength:%d,\t   length: %d,\t   leavesCount:%d ,\n", 
            i, j, d,splitPos, minCommonPrefixLength, prefixLength, maxLength,length,leavesCount);
    }
    */

    internals[i].visited = 0;
}

__global__ 
void computeBounds(int leavesCount, BVHLeafNode* leaves, BVHInternalNode* internals){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= leavesCount) return;

    int curr = leaves[i].parent;
    //printf("this is leaf %d, next is %d\n", i,curr);
    while(atomicInc(&(internals[curr].visited),2) > 0){
        //printf("doing bounds for node %d\n", curr);
        AABB boundsLeft;
        AABB boundsRight;
        BVHInternalNode& node = internals[curr];
        if(node.leftChildIsLeaf){
            boundsLeft = leaves[node.leftChild].box;
        }
        else{
            boundsLeft = internals[node.leftChild].box;
        }

        if(node.rightChildIsLeaf){
            boundsRight = leaves[node.rightChild].box;
        }
        else{
            boundsRight = internals[node.rightChild].box;
        }

        node.box = unionBoxes(boundsLeft,boundsRight);
        if (curr == 0) {
            break;
        } 

        curr = node.parent;
    }
}

__device__
void copyLeafNode(BVHLeafNode& leaf, BVHNode& node){
    node.box = leaf.box;
    node.isLeaf = true;
    node.primitiveIndex = leaf.primitiveIndex;
}

__global__ 
void mergeNodesArray(int leavesCount, BVHLeafNode* leaves, BVHInternalNode* internals, BVHNode* nodes){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= leavesCount-1) return;

    nodes[i].isLeaf = false;

    nodes[i].box = internals[i].box;

    if(internals[i].leftChildIsLeaf){
        int leftChildLeaf = leavesCount - 1 + internals[i].leftChild;
        nodes[i].leftChild = leftChildLeaf;
        copyLeafNode(leaves[internals[i].leftChild],nodes[leftChildLeaf]);
    }
    else{
        nodes[i].leftChild = internals[i].leftChild;
    }

    if(internals[i].rightChildIsLeaf){
        int rightChildLeaf = leavesCount - 1 + internals[i].rightChild;
        nodes[i].rightChild = rightChildLeaf;
        copyLeafNode(leaves[internals[i].rightChild],nodes[rightChildLeaf]);
    }
    else{
        nodes[i].rightChild = internals[i].rightChild;
    }
}

BVH BVH::build(Triangle* primitivesDevice, int primitivesCount,const AABB& sceneBounds){
    std::cout << "started building bvh" << std::endl;
    BVH bvh(primitivesCount);

    GpuArray<BVHLeafNode> leaves(primitivesCount);

    
    int numThreadsPrimitives = min(primitivesCount,MAX_THREADS_PER_BLOCK);
    int numBlocksPrimitives = divUp(primitivesCount,numThreadsPrimitives);
    fillLeafBoundingBoxes <<< numBlocksPrimitives, numThreadsPrimitives >>> (primitivesDevice,primitivesCount, leaves.data);
    CHECK_IF_CUDA_ERROR("fill leaf bounding boxes");

    GpuArray<unsigned int> leafMortonCodes(primitivesCount);
    fillLeafMortonCodes<<< numBlocksPrimitives, numThreadsPrimitives >>> (primitivesDevice,primitivesCount,leaves.data, leafMortonCodes.data,sceneBounds);
    CHECK_IF_CUDA_ERROR("fill leaf morton");

    thrust::stable_sort_by_key(thrust::device, leafMortonCodes.data,leafMortonCodes.data+primitivesCount,leaves.data,thrust::less<unsigned int>());

    GpuArray<BVHInternalNode> internals(primitivesCount-1);

    int numThreadsInternals = min(primitivesCount-1,MAX_THREADS_PER_BLOCK);
    int numBlocksInternals = divUp(primitivesCount-1,numThreadsPrimitives);
    buildRadixTree <<< numBlocksInternals, numThreadsInternals >>> (primitivesCount,leafMortonCodes.data, leaves.data,internals.data);
    CHECK_IF_CUDA_ERROR("build radix tree");

    computeBounds <<< numBlocksPrimitives, numThreadsPrimitives >>> (primitivesCount,leaves.data,internals.data);
    CHECK_IF_CUDA_ERROR("compute bounds");


    mergeNodesArray <<< numBlocksInternals, numThreadsInternals >>> (primitivesCount,leaves.data,internals.data, bvh.nodes.gpu.data);
    CHECK_IF_CUDA_ERROR("merge nodes array");    

    return bvh;
}
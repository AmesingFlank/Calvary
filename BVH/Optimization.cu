#include "hip/hip_runtime.h"
#include "../Utils/GpuCommons.h"
#include "Optimization.h"
#include "../Utils/Array.h"
#include "../Utils/Utils.h"
#include <vector>

std::vector<unsigned int> scheduleCPU = {

/*round 1*/
0b00011,0b00110,0b00101,0b01001,0b01010,0b01100,0b10001,0b10010,0b10100,0b11000, /*2bits; greatest at 5 or less*/ 
0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0, /*round 1 vacancies*/

/*round 2*/
0b00111,0b01011,0b01101,0b01110,0b10011,0b10101,0b10110,0b11001,0b11010,0b11100,/*3bits; greatest at 5 or less*/     
0b100001,0b100010,0b100100,0b101000,0b110000,/*2 bits; 6th bit set*/  
0b1000001,0b1000010,0b1000100,0b1001000,0b1010000,0b1100000/*2 bits; 7th bit set*/, 
0,0,0,0,0,0,0,0,0,0,0, /*round 2 vacancies*/


/*round 3*/
0b100011,0b100101,0b100110,0b101001,0b101010,0b101100,0b110001,0b110010,0b110100,0b111000, /*3bits; 6th set*/
0b1000011,0b1000101,0b1000110,0b1001001,0b1001010,0b1001100,0b1010001,0b1010010,0b1010100,0b1011000,0b1100001,0b1100010,0b1100100,0b1101000,0b1110000,/*3bits; 7th set*/
0b0001111,0b0010111,0b0011110, /*some of 4bits*/
0,0,0,0, /*round 3 vacancies*/


/*round 4; all the remaining 4 bits*/
0b1000111,0b0011011,0b0101011,0b1001011,0b0110011,0b1010011,0b1100011,0b0011101,0b0101101,0b1001101,0b0110101,0b1010101,0b1100101,0b0111001,0b1011001,0b1101001,0b1110001,0b0100111,0b0101110,0b1001110,0b0110110,0b1010110,0b1100110,0b0111010,0b1011010,0b1101010,0b1110010,0b0111100,0b1011100,0b1101100,0b1110100,0b1111000,

/*round 5; all the 5 bits*/
0b0011111,0b0101111,0b1001111,0b0110111,0b1010111,0b1100111,0b0111011,0b1011011,0b1101011,0b1110011,0b0111101,0b1011101,0b1101101,0b1110101,0b1111001,0b0111110,0b1011110,0b1101110,0b1110110,0b1111010,0b1111100,
0,0,0,0,0,0,0,0,0,0,0

};

__global__
void tryCollapseNodes(int nodesCount, BVHNode* nodes, unsigned int* visited){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= nodesCount) return;

    if(!nodes[i].isLeaf){
        return;
    }
    int curr = nodes[i].parent;

    while(atomicInc(&(visited[curr]),2) > 0){
        BVHNode& node = nodes[curr];
      
        BVHNode& leftChild = nodes[node.leftChild];
        BVHNode& rightChild = nodes[node.rightChild];

        if(leftChild.isLeaf && rightChild.isLeaf){
            constexpr float primCost = 1;
            constexpr float boxCost = 0.0006;
            
            float originalCost =
                (leftChild.primitivesCount() * primCost * leftChild.surfaceArea/node.surfaceArea) + boxCost+
                (rightChild.primitivesCount() * primCost * rightChild.surfaceArea / node.surfaceArea) + boxCost;

            float temp = leftChild.surfaceArea / node.surfaceArea + rightChild.surfaceArea / node.surfaceArea;
            
            float collapsedCost = (leftChild.primitivesCount() + rightChild.primitivesCount()) * primCost;

            if(collapsedCost <  originalCost ){
                node.isLeaf = true;
                node.primitiveIndexBegin = leftChild.primitiveIndexBegin;
                node.primitiveIndexEnd = rightChild.primitiveIndexEnd;
                continue;
            }
        }
        return;
    }
}


void collapseNodes(int primitivesCount,GpuArray<BVHNode>& nodes){
    GpuArray<unsigned int> visited(nodes.N);
    int numBlocks,numThreads;
    setNumBlocksThreads(nodes.N, numBlocks, numThreads);
    tryCollapseNodes<<<numBlocks,numThreads>>> (nodes.N,nodes.data,visited.data);
    CHECK_IF_CUDA_ERROR("collapseNodes");
}


void optimizeBVH(int primitivesCount,GpuArray<BVHNode>& nodes){

}
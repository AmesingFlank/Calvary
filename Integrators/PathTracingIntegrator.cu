#include "hip/hip_runtime.h"
#include "PathTracingIntegrator.h"
#include "../Samplers/SimpleSampler.h"
#include "../Utils/TaskQueue.h"



namespace PathTracing {

    PathTracingIntegrator::PathTracingIntegrator(int maxDepth_):maxDepth(maxDepth_) {

    }

    struct MaterialEvalTask {
        IntersectionResult intersection;
        Ray rayToLight;
        Spectrum incident;
        Ray exitantRay;
        float multiplier;
        Spectrum* result;
    };



    __device__
    static void renderRay(const SceneHandle& scene, const Ray& ray, SamplerObject& sampler, Spectrum* result, TaskQueue<MaterialEvalTask>& materialEvalQueue,int maxDepth) {
        
        *result = make_float3(0,0,0);
        Spectrum multiplier = make_float3(1, 1, 1);

        Ray thisRay = ray;

        sampler.startPixel();
        

        for (int i = 0; i < maxDepth; ++i) {
            IntersectionResult intersection;
            scene.intersect(intersection, thisRay);

            if (!intersection.intersected) {
                if (scene.hasEnvironmentMap()) {
                    *result += scene.getEnvironmentMap()->EnvironmentMap::evaluateRay(thisRay)*multiplier;
                }
                return;
            }


            
            const Primitive* prim = intersection.primitive;

            if (prim->areaLight && i == 0) {
                *result += prim->areaLight->get<DiffuseAreaLight>()->DiffuseAreaLight::evaluateRay(thisRay)*multiplier;
            }


            Ray exitantRay = { intersection.position,thisRay.direction * -1 };

            int lightIndex = sampler.randInt(scene.lightsCount);

            const LightObject& light = scene.lights[lightIndex];
            Ray rayToLight;
            float probability;
            float4 randomSource = sampler.rand4();

            VisibilityTest visibilityTest;
            visibilityTest.sourceGeometry = prim->shape.getID();
            
            
            Spectrum incident = light.sampleRayToPoint(intersection.position, randomSource, probability, rayToLight, visibilityTest);

            if (scene.testVisibility(visibilityTest) && dot(rayToLight.direction, intersection.normal) > 0) {
                if (probability == 0) {
                    printf("probability is 0\n");
                }
                //MaterialEvalTask task = { intersection,rayToLight,incident,exitantRay,(float)scene.lightsCount / probability,result };
                //materialEvalQueue.push(task);
                *result += prim->material.eval(rayToLight, incident, exitantRay, intersection) * scene.lightsCount*multiplier / probability;
            }

            Ray nextRay;
            float nextRayProbability;
            Spectrum nextMultiplier = prim->material.getBSDF().sample(sampler.rand2(), nextRay.direction, thisRay.direction * -1.f, &nextRayProbability);
            nextRay.origin = intersection.position + nextRay.direction * 0.0001f;
            thisRay = nextRay;
            multiplier = multiplier * nextMultiplier * abs(dot(nextRay.direction,intersection.normal)) / nextRayProbability;



        }
        
        

    }


    __global__
    void renderAllSamples(CameraSample* samples, int samplesCount, SceneHandle scene, CameraObject camera, SamplerObject sampler, Spectrum* results, TaskQueue<MaterialEvalTask> materialEvalQueue,int maxDepth) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= samplesCount) {
            return;
        }

        Ray ray = camera.genRay(samples[index]);
        Spectrum* result = &results[index];

        renderRay(scene, ray, sampler, result, materialEvalQueue,maxDepth);

    }


    __device__
    void runMaterialEval(MaterialEvalTask& task) {
        const Primitive* prim = task.intersection.primitive;

        *(task.result) += prim->material.eval(task.rayToLight, task.incident, task.exitantRay, task.intersection) * task.multiplier;
    }


    __global__
    void addSamplesToFilm(FilmObject film, Spectrum* result, CameraSample* samples, int count) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= count) {
            return;
        }
        film.addSample(samples[index], result[index]);
    }



    RenderResult PathTracingIntegrator::render(const Scene& scene, const CameraObject& camera, FilmObject& film) {

        GpuArray<CameraSample> allSamples = sampler->genAllCameraSamples(camera, film);

        SceneHandle sceneHandle = scene.getDeviceHandle();

        SamplerObject& samplerObject = *sampler;


        int samplesCount = (int)allSamples.N;
        int numThreads = min(samplesCount, MAX_THREADS_PER_BLOCK);
        int numBlocks = divUp(samplesCount, numThreads);

        sampler->prepare(samplesCount);

        GpuArray<Spectrum> result(samplesCount);
        TaskQueue<PathTracing::MaterialEvalTask> materialEvalQueue(samplesCount);

        CHECK_IF_CUDA_ERROR("before render all samples");
        PathTracing::renderAllSamples << <numBlocks, numThreads >> >
            (allSamples.data, samplesCount, sceneHandle, camera, samplerObject.getCopyForKernel(), result.data, materialEvalQueue.getCopyForKernel(),maxDepth);
        CHECK_IF_CUDA_ERROR("render all samples");

        
        materialEvalQueue.forAll(
            [] __device__
            (PathTracing::MaterialEvalTask & task) {
            PathTracing::runMaterialEval(task);
        }
        );

        PathTracing::addSamplesToFilm << <numBlocks, numThreads >> > (film.getCopyForKernel(), result.data, allSamples.data, samplesCount);
        CHECK_CUDA_ERROR("add sample to film");


        return film.readCurrentResult();
    }

}

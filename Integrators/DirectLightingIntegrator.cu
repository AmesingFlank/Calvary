#include "hip/hip_runtime.h"
#include "DirectLightingIntegrator.h"
#include "../Samplers/SimpleSampler.h"
#include "../Utils/TaskQueue.h"
#include "../Core/Impl.h"


namespace DirectLighting {

    DirectLightingIntegrator::DirectLightingIntegrator() {

    }

    struct MaterialEvalTask {
        IntersectionResult intersection;
        Ray rayToLight;
        Spectrum incident;
        Ray exitantRay;
        float multiplier;
        Spectrum* result;
    };
    
    __device__
    static void renderRay(const SceneHandle& scene, const Ray& ray, SamplerObject& sampler, Spectrum* result, TaskQueue<MaterialEvalTask>& materialEvalQueue) {
        IntersectionResult intersection;
        scene.intersect(intersection, ray);
    
        *result = make_float3(0, 0, 0);
       
        sampler.startPixel();
        if (!intersection.intersected) {
            if (scene.hasEnvironmentMap()) {
                *result = scene.getEnvironmentMap()->EnvironmentMap::evaluateRay(ray);
            }
            return;
        }
        
    
    
        const Primitive* prim = intersection.primitive;
    
        
    
        if (prim->areaLight) {
            *result += prim->areaLight->get<DiffuseAreaLight>()->DiffuseAreaLight::evaluateRay(ray);
        }
    
        
    
        Ray exitantRay = { intersection.position,ray.direction * -1 };
    
        int lightIndex = sampler.randInt(scene.lightsCount);
    
    
        const LightObject& light = scene.lights[lightIndex];
        Ray rayToLight;
        float probability;
        float4 randomSource = sampler.rand4();
    
        VisibilityTest visibilityTest;
        visibilityTest.sourceTriangleIndex = intersection.triangleIndex;
    
    
        Spectrum incident = light.sampleRayToPoint(intersection.position, sampler, probability, rayToLight, visibilityTest);


        if (scene.testVisibility(visibilityTest)) {
            if (probability == 0) {
                printf("probability is 0\n");
            }
            MaterialEvalTask task = {intersection,rayToLight,incident,exitantRay,(float)scene.lightsCount/probability,result};
            materialEvalQueue.push(task);
            //result += prim->material.eval(rayToLight, incident, exitantRay, intersection) * scene.lightsCount / probability;
        }
    
    }
    
    
    __global__
    void renderAllSamples(CameraSample* samples, int samplesCount, SceneHandle scene, CameraObject camera, SamplerObject sampler, Spectrum* results,TaskQueue<MaterialEvalTask> materialEvalQueue) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= samplesCount) {
            return;
        }
    
        Ray ray = camera.genRay(samples[index]);
        Spectrum* result = &results[index];
    
        renderRay(scene, ray, sampler,result,materialEvalQueue);
    
    }
    
    
    __device__
    void runMaterialEval(MaterialEvalTask& task) {
        const Primitive* prim = task.intersection.primitive;
    
        *(task.result) += prim->material.eval(task.rayToLight, task.incident, task.exitantRay, task.intersection) * task.multiplier;
    }
    
    
    __global__
    void addSamplesToFilm(FilmObject film, Spectrum* result,CameraSample* samples, int count) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= count) {
            return;
        }
        film.addSample(samples[index], result[index]);
    }



    void DirectLightingIntegrator::render(const Scene& scene, const CameraObject& camera, FilmObject& film) {

        while(!isFinished(scene,camera,film)){
            GpuArray<CameraSample> allSamples = sampler->genAllCameraSamples(camera, film);

            SceneHandle sceneHandle = scene.getDeviceHandle();

            SamplerObject& samplerObject = *sampler;


            int samplesCount = (int)allSamples.N;
            int numThreads = min(samplesCount, MAX_THREADS_PER_BLOCK);
            int numBlocks = divUp(samplesCount, numThreads);

            sampler->prepare(samplesCount);

            GpuArray<Spectrum> result(samplesCount);
            TaskQueue<DirectLighting::MaterialEvalTask> materialEvalQueue(samplesCount);

            CHECK_IF_CUDA_ERROR("before render all samples");
            DirectLighting::renderAllSamples << <numBlocks, numThreads >> >
                (allSamples.data, samplesCount, sceneHandle, camera, samplerObject.getCopyForKernel(), result.data, materialEvalQueue.getCopyForKernel());
            CHECK_IF_CUDA_ERROR("render all samples");

            materialEvalQueue.forAll(
                [] __device__
                (DirectLighting::MaterialEvalTask & task) {
                DirectLighting::runMaterialEval(task);
            }
            );

            DirectLighting::addSamplesToFilm << <numBlocks, numThreads >> > (film.getCopyForKernel(), result.data, allSamples.data, samplesCount);
            CHECK_CUDA_ERROR("add sample to film");
        }

        
    }
    
}

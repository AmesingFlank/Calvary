#include "hip/hip_runtime.h"
#include "DirectLightingCPUIntegrator.h"
#include "../Samplers/SimpleSamplerCPU.h"

DirectLightingCPUIntegrator::DirectLightingCPUIntegrator() {
    sampler = std::make_unique<SamplerObject>(SimpleSamplerCPU());
}

Spectrum DirectLightingCPUIntegrator::renderRay(const SceneHandle& scene, const Ray& ray, SamplerObject& sampler){

    IntersectionResult intersection;
    scene.intersect(intersection,ray);

    
    if(!intersection.intersected){
        if (scene.hasEnvironmentMap()) {
            return scene.getEnvironmentMap()->EnvironmentMap::evaluateRay(ray);
        }
        return make_float3(0, 0, 0);
    }
    
    
    Spectrum result = make_float3(0,0,0);

	const Primitive* prim = intersection.primitive;

    Ray exitantRay = {intersection.position,ray.direction*-1};

    for (int i = 0; i < scene.lightsCount;++i) {
        const LightObject& light = scene.lights[i];
        Ray rayToLight;
        float probability;
        float4 randomSource = sampler.rand4();

        VisibilityTest visibilityTest;
        visibilityTest.sourceGeometry = prim->shape.getID();

        Spectrum incident = light.sampleRayToPoint(intersection.position, randomSource, probability, rayToLight,visibilityTest);
        if(scene.testVisibility(visibilityTest) && dot(rayToLight.direction, intersection.normal) > 0){
            result += prim->material.eval(rayToLight,incident,exitantRay,intersection) / probability;
        }
    }
    return result;
}
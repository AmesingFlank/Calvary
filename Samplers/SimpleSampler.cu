#include "hip/hip_runtime.h"
#include "SimpleSampler.h"
#include "../Utils/GpuCommons.h"

#include "../Utils/RandomUtils.h"
#include <iostream>


SimpleSampler::SimpleSampler(int samplesPerPixel_, bool isCopyForKernel_ ):states(1024,isCopyForKernel_),samplesPerPixel(samplesPerPixel_){
    
}

SimpleSampler::SimpleSampler() :states(0,true) {

}

SimpleSampler SimpleSampler::getCopyForKernel(){
    SimpleSampler copy(samplesPerPixel,true);
    copy.states = states.getCopyForKernel();
    return copy;
}


__global__
void genNaiveSample(CameraSample* resultPointer, int samplesCount, int width, int height,int samplesPerPixel,SimpleSampler sampler){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= samplesCount){
        return;
    }

    int pixelIndex = index / samplesPerPixel;

    int x = pixelIndex % width;
    int y = pixelIndex / width;

    CameraSample sample{ (float)x + 0.5 , (float)y +0.5 };
    sample.x += 0.5*sampler.rand1() - 0.25;
    sample.y += 0.5*sampler.rand1() - 0.25;

    resultPointer[index] = sample;
}


GpuArray<CameraSample> SimpleSampler::genAllCameraSamples(const CameraObject& camera, FilmObject& film) {
    int width = film.getWidth();
    int height = film.getHeight();
    int count = width*height * samplesPerPixel;

    std::cout << "about to alloc cam samples " << samplesPerPixel << std::endl;

    GpuArray<CameraSample> result(count);

    int numThreads = min(count,MAX_THREADS_PER_BLOCK);
    int numBlocks = divUp(count,numThreads);

    genNaiveSample <<<numBlocks,numThreads>>> (result.data,count,width,height,samplesPerPixel,getCopyForKernel());
    CHECK_IF_CUDA_ERROR("gen naive samples");
    return result;
}
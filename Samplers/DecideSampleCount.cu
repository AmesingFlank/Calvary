#include "hip/hip_runtime.h"
#include "DecideSampleCount.h"
#include "../Utils/GpuCommons.h"


// we might not have enough memory to render all samples at once.



int decideSamplesPerPixel(FilmObject& film, int totalSPP,int bytesNeededPerSample,int maxSamplesPerRound){
    int sppLeftToDo = totalSPP - film.getCompletedSamplesPerPixel();
    int resolution = film.getWidth() * film.getHeight();

    
    size_t freeMemory, totalMemory;
    HANDLE_ERROR(hipMemGetInfo(&freeMemory, &totalMemory));
    size_t freeMemoryPerPixel = freeMemory / resolution;
    printf("free memory per pixel %d \n", freeMemoryPerPixel);
    
    int thisSPP = min(sppLeftToDo,(int)(freeMemoryPerPixel / bytesNeededPerSample));
    if (maxSamplesPerRound != -1) {
        thisSPP = min(thisSPP, maxSamplesPerRound);
    }
    
    printf("SPP left:%d,  SPP this round:%d,   SPP completed: %d \n", sppLeftToDo, thisSPP, film.getCompletedSamplesPerPixel());

    film.setCompletedSamplesPerPixel(film.getCompletedSamplesPerPixel() + thisSPP);
    return thisSPP;
}
#include "hip/hip_runtime.h"
#include "DecideSampleCount.h"
#include "../Utils/GpuCommons.h"


// we might not have enough memory to render all samples at once.


#define MAX_SAMPLES_PER_RUN 1024*1024*4


int decideSamplesPerPixel(FilmObject& film, int totalSPP){
    int sppLeftToDo = totalSPP - film.getCompletedSamplesPerPixel();
    int resolution = film.getWidth() * film.getHeight();

    
    size_t freeMemory, totalMemory;
    HANDLE_ERROR(hipMemGetInfo(&freeMemory, &totalMemory));
    size_t freeMemoryPerPixel = freeMemory / resolution;
    printf("free memory per pixel %d \n", freeMemoryPerPixel);
    
    int thisSPP = min(sppLeftToDo,(int)(freeMemoryPerPixel / 900));
    
    printf("SPP left:%d,  SPP this round:%d,   SPP completed: %d \n", sppLeftToDo, thisSPP, film.getCompletedSamplesPerPixel());

    film.setCompletedSamplesPerPixel(film.getCompletedSamplesPerPixel() + thisSPP);
    return thisSPP;
}
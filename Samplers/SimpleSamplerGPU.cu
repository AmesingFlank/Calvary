#include "hip/hip_runtime.h"
#include "SimpleSamplerGPU.h"
#include "../Utils/GpuCommons.h"

#include "../Utils/RandomUtils.h"
#include <iostream>


SimpleSamplerGPU::SimpleSamplerGPU(int samplesPerPixel_, bool isCopyForKernel_ ):states(1024,isCopyForKernel_),samplesPerPixel(samplesPerPixel_){
    
}

SimpleSamplerGPU::SimpleSamplerGPU() :states(0,true) {

}

SimpleSamplerGPU SimpleSamplerGPU::getCopyForKernel(){
    SimpleSamplerGPU copy(samplesPerPixel,true);
    copy.states = states.getCopyForKernel();
    return copy;
}


__global__
void genNaiveSample(CameraSample* resultPointer, int samplesCount, int width, int height,int samplesPerPixel,SimpleSamplerGPU sampler){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= samplesCount){
        return;
    }

    int pixelIndex = index / samplesPerPixel;

    int x = pixelIndex % width;
    int y = pixelIndex / width;

    CameraSample sample{ (float)x + 0.5 , (float)y +0.5 };
    sample.x += 0.5*sampler.rand1() - 0.25;
    sample.y += 0.5*sampler.rand1() - 0.25;

    resultPointer[index] = sample;
}


GpuArray<CameraSample> SimpleSamplerGPU::genAllCameraSamples(const CameraObject& camera, FilmObject& film) {
    int width = film.getWidth();
    int height = film.getHeight();
    int count = width*height * samplesPerPixel;

    std::cout << "about to alloc cam samples " << samplesPerPixel << std::endl;

    GpuArray<CameraSample> result(count);

    int numThreads = min(count,MAX_THREADS_PER_BLOCK);
    int numBlocks = divUp(count,numThreads);

    genNaiveSample <<<numBlocks,numThreads>>> (result.data,count,width,height,samplesPerPixel,getCopyForKernel());
    CHECK_IF_CUDA_ERROR("gen naive samples");
    return result;
}
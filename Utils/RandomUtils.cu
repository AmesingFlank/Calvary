#include "hip/hip_runtime.h"
#include "RandomUtils.h"

__global__ void initCurandStates ( hiprandState * states, unsigned long seed, int maxThreads )
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= maxThreads){
        return;
    }

    hiprand_init ( seed, index, 0, &states[index] );
}



/*
__global__ void initSobolCurandStates ( hiprandStateSobol32 * states,int N, unsigned int* directionVectors){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= N){
        return;
    }

    hiprand_init ( directionVectors, index, &states[index] );
}
*/
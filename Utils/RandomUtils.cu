#include "hip/hip_runtime.h"
#include "RandomUtils.h"

__global__ void initCurandStates ( hiprandState * states, unsigned long seed, int maxThreads )
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= maxThreads){
        return;
    }

    hiprand_init ( seed, index, 0, &states[index] );
}

